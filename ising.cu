#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 16

//! Perform the Ising simulation
//! @param lattice A pointer to the lattice of atoms.
//! @param height The number of rows of the input lattice.
//! @param width The number of columns of the input lattice.
//! @param T The temperature, in units of epsilon/k. Epsilon is the exchange energy and k is the boltzmann constant.
//! #param iterations The number of Metropolis iterations to perform.
__global__ void ising(int * lattice, int height, int width, float T, unsigned int iterations) {
	__shared__ int slattice[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int sneighbors[4 * BLOCK_SIZE];

	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int deltaU, top, bottom, left, right;
	hiprandState_t state;

	//Initialize random number generator
	/* From the hiprand library guide:
	   Each experiment should be assigned a unique seed.
	   Each thread should have a unique sequenc number. 
	*/
	//TODO: Add seed and sequence
	hiprand_init(0, 0, 0, &state); //seed, sequence, offset

	//Load sublattice into shared memory
	if(tx < width && ty < height) {
		slattice[threadIdx.y][threadIdx.x] = lattice[ty * width + threadIdx.x];
		if(threadIdx.y == 0) {
			if(ty == 0)
				sneighbors[threadIdx.x] = lattice[(height - 1) * width + tx];
			else
				sneighbors[threadIdx.x] = lattice[(ty - 1) * width + tx];
		}

		if(threadIdx.y == (BLOCK_SIZE - 1)) {
			if(ty == (height - 1))
				sneighbors[2 * BLOCK_SIZE + threadIdx.x] = lattice[tx];
			else
				sneighbors[2 * BLOCK_SIZE + threadIdx.x] = lattice[(ty + 1) * width + tx];
		}

		if(threadIdx.x == 0) {
			if(tx == 0)
				sneighbors[3 * BLOCK_SIZE + threadIdx.y] = lattice[ty * width + (width - 1)];
			else
				sneighbors[3 * BLOCK_SIZE + threadIdx.y] = lattice[ty * width + (tx - 1)];
		}

		if(threadIdx.x == (BLOCK_SIZE - 1)) {
			if(tx == (width - 1))
				sneighbors[BLOCK_SIZE + threadIdx.y] = lattice[ty * width];
			else
				sneighbors[BLOCK_SIZE + threadIdx.y] = lattice[ty * width + (tx + 1)];
		}
		__syncthreads();

		//Perform simulation
		//Each turn of the loop performs BLOCK_SIZE^2 iterations of the Metropolis algorithm
		for(int k = 0; k < iterations; k += (BLOCK_SIZE * BLOCK_SIZE)) {
			for(int i = 0; i < 2; i ++) {
				//Checkerboard
				if((threadIdx.x + threadIdx.y) % 2 == i) {
					if(threadIdx.y == 0)
						top = sneighbors[threadIdx.x]; 
					else
						top = slattice[threadIdx.y - 1][threadIdx.x];
					
					if(threadIdx.x == 0)
						left = sneighbors[3 * BLOCK_SIZE + threadIdx.y];
					else
						left = slattice[threadIdx.y][threadIdx.x - 1];

					if(threadIdx.y == (BLOCK_SIZE - 1))
						bottom = sneighbors[2 * BLOCK_SIZE + threadIdx.x]
					else
						bottom = slattice[threadIdx.y + 1][threadIdx.x];

					if(threadIdx.x == (BLOCK_SIZE - 1))
						right = sneighbors[BLOCK_SIZE + threadIdx.y]; 
					else
						right = slattice[threadIdx.y][threadIdx.x + 1];

					//Calculate change in energy if dipole were flipped
					deltaU = 2 * slattice[threadIdx.y][threadIdx.x] * (top + bottom + left + right);

					//If the energy would decrease, flip the dipole
					if(deltaU <= 0)
						slattice[threadIdx.y][threadIdx.x] *= -1;
					else {
						rand = hiprand_uniform(&state);
						//TODO: Put in a more precise value of e
						//Else the probability of a flip is given by the Boltzmann factor
						if(rand < powf(2.71f, -deltaU/T))
							slattice[threadIdx.y][threadIdx.x] *= -1;
					}
					__syncthreads(); //Is this needed?
				}
			}
		}
	slattice[threadIdx.y][threadIdx.x] = lattice[ty * width + threadIdx.x];
	}

}

//! Generate an image file of the lattice and write it to disk
//! @param lattice A 2D array of ints, whose values are either 1 or -1.
//! @param height The number of rows of the lattice
//! @param width The number of colums of the lattice
//! @param filename The name of the file to write to.
void print(int * lattice, int height, int width, char * filename) {

}
